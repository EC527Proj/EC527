#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define BLOCK_SIZE				16
#define WIDTH					1024
#define NUM_BLOCKS				WIDTH / 16
#define TILE_WIDTH				16

#define GIG 					1000000000
#define CPG 					3.6

#define PRINT_TIME 				1
#define SM_ARR_LEN				128
#define TOL						11


#define OMEGA 					1.90
#define MINVAL   				0.0
#define MAXVAL  				10.0

#define IMUL(a, b) __mul24(a, b)

void host_bb(float * arr);
void initializeArray1D(float *arr, int len, int seed);
void SOR(float * v, int arrLen);
void print_matrix(float * v, int len);
struct timespec diff(struct timespec start, struct timespec end);

__global__ void mmm_global( float *Md, float *Nd, float *Od ) {
/*
	
	*/
	const int tid = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int threadN = IMUL(blockDim.x, gridDim.x);
	
	int i;
	
	for(i = tid; i < WIDTH; i += threadN) {
		Od[i] = Md[i] * 2;
	}
}

__global__ void bubblesort_d(float* scratch_global)  { 
    int offset = (blockDim.x*blockIdx.x+threadIdx.x)*numchn;
    int k1, k2, i;
    float temp1;
    float scratch[numchn]; //local memory
    for(i=0;i<numchn;i++)
        scratch[i]=scratch_global[offset+i];
    /* Perform a bubble sort */
    for (k1=0; k1<numchn-1; k1++) {
        for (k2=0; k2<numchn-1-k1; k2++) {
            if (scratch[k2]>scratch[k2+1]) {
                temp1 = scratch[k2];
                scratch[k2]=scratch[k2+1];
                scratch[k2+1]=temp1;
    }   }   }
    //__syncthreads();        
    for(i=0;i<numchn;i++)
        scratch_global[offset+i] = scratch[i];
}


int main(int argc, char **argv) {
	int arrLen = 0, i, j;
		
	// GPU Timing variables
	hipEvent_t start, stop;
	float elapsed_gpu;
	
	//CPU timing variables
	struct timespec time1, time2, time_elapsed;
	
	
	// Arrays on GPU global memory
	float *d_m;
	float *d_n;
	float *d_o1;
	float *d_o2;
	float *d_o3;

	// Arrays on the host memory
	float *h_m;
	float *h_n;
	float *h_o1;
	float *h_o2;
	float *h_o3;
	float *h_gold;
	
	
	if (argc > 1) {
		arrLen  = atoi(argv[1]);
	}
	else {
		arrLen = WIDTH;
	}

	printf("Size of the matrix = %dx%d\n", arrLen, arrLen);

	// Allocate GPU memory
	size_t allocSize = WIDTH * WIDTH * sizeof(float);
	CUDA_SAFE_CALL(hipMalloc((void **)&d_m, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_n, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_o1, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_o2, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_o3, allocSize));
		
	// Allocate arrays on host memory
	h_m                   = (float *) malloc(allocSize);
	h_n                   = (float *) malloc(allocSize);
	h_o1                   = (float *) malloc(allocSize);
	h_o2                   = (float *) malloc(allocSize);
	h_o3                   = (float *) malloc(allocSize);
	h_gold                 = (float *) malloc(allocSize);
	
	// Initialize the host arrays
	printf("\nInitializing the arrays ...");
	// Arrays are initialized with a known seed for reproducability
	initializeArray1D(h_m, WIDTH, 2453);
	initializeArray1D(h_n, arrLen, 2453);
	initializeArray1D(h_gold, arrLen, 2453);
	printf("\t... done\n\n");
	
	//Make the destination matrix all zeros 
	for(i = 0; i < WIDTH; i++)
		for(j = 0; j < WIDTH; j++) {
			h_o1[i + WIDTH + j] = 0.0;
			h_o2[i + WIDTH + j] = 0.0;
			h_o3[i + WIDTH + j] = 0.0;
		
		}
		
//mmm_global****************************************	
printf("\n*****************************\nmm_global");

	// Transfer the arrays to the GPU memory
	CUDA_SAFE_CALL(hipMemcpy(d_m, h_m, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_n, h_n, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_o1, h_o1, allocSize, hipMemcpyHostToDevice));
	  
	dim3 myBlock1(NUM_BLOCKS, NUM_BLOCKS);
	dim3 myThreads1(WIDTH / myBlock1.x, WIDTH / myBlock1.y);
#if PRINT_TIME
	// Create the cuda events
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Record event on the default stream
	hipEventRecord(start, 0);
#endif	
	// Launch the kernel 
	mmm_global<<<16, 256>>>(d_m, d_n, d_o1);
#if PRINT_TIME
	// Stop and destroy the timer
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	printf("\nGPU time: %f (msec)\n", elapsed_gpu);
	hipEventDestroy(start);
	hipEventDestroy(stop);
#endif
	
	// Transfer the results back to the host
	CUDA_SAFE_CALL(hipMemcpy(h_o1, d_o1, allocSize, hipMemcpyDeviceToHost));
	
//mm_host****************************************	
printf("\n*****************************\nmm_host");
	// Compute the results on the host
	clock_gettime(CLOCK_REALTIME, &time1);
	
	host_bb(h_m, h_n, h_gold);
	clock_gettime(CLOCK_REALTIME, &time2);
	
	time_elapsed = diff(time1,time2);
	printf("\nCPU time: %ld (nsec)\n", (long int)((double)(CPG)*(double)(GIG * time_elapsed.tv_sec + time_elapsed.tv_nsec)) ); 
	
	printf("Calculations done.\n");
	

//Checking mmm_global****************************************	
printf("\n*****************************\nChecking mmm_global...");	
	int errCount = 0, zeroCount = 0;
	// Compare the results
	for(i = 0; i < WIDTH; i++) {
		if (abs(h_gold[i] - h_o1[i]) > TOL) {
			errCount++;
			printf("Failure at [%d][%d]: %.4f vs %.4f\n",i,j,h_o1[i],h_gold[i]);
		}
		if (h_gold[i] == 0) {
			zeroCount++;
			
		}
	} 
	
	if (errCount > 0) {
		printf("@ERROR: TEST FAILED: %d results did not matched\n", errCount);
	}
	else if (zeroCount > 0){
		printf("@ERROR: TEST FAILED: %d results (from GPU) are zero\n", zeroCount);
	}
	else {
		printf("...TEST PASSED: All results matched\n");
	}

	// Free-up device and host memory
	//CUDA_SAFE_CALL(hipFree(d_P2));
	//CUDA_SAFE_CALL(hipFree(d_P3));
	
	hipFree(d_m);
	hipFree(d_n);
	hipFree(d_o1);
	hipFree(d_o2);
	hipFree(d_o3);
	free(h_m);
	free(h_n);
	free(h_o1);
	free(h_o2);
	free(h_o3);
	free(h_gold);
		
	return 0;
}

void host_bb(float * arr) {
	bool swapped = true;
	int j = 0;
	int tmp;
	while (swapped) {
		swapped = false;
		j++;
		for (int i = 0; i < n - j; i++) {
			if (arr[i] > arr[i + 1]) {
				tmp = arr[i];
				arr[i] = arr[i + 1];
				arr[i + 1] = tmp;
				swapped = true;
			}
		}
	}

}

void initializeArray1D(float *arr, int len, int seed) {
	int i;
	float fRand(float fMin, float fMax);
	srand(seed);

	for (i = 0; i < len; i++) {
			arr[i] = (fRand((float)(MINVAL),(float)(MAXVAL)));
		
	}
}
 
float fRand(float fMin, float fMax)
{
    float f = (float)random() / RAND_MAX;
    return fMin + f * (fMax - fMin);
}

/* print vector */
void print_matrix(float * v, int len)
{
  long int i;

  printf("\n length = %ld", len);
  for (i = 0; i < len; i++) {
  }
}

struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}
